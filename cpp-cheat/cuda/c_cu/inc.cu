
#include <hip/hip_runtime.h>
/* Minimal example. Increment a vector. */

#define N 3

__global__ void inc(int *a) {
    int i = blockIdx.x;
    if (i<N) {
        a[i]++;
    }
}
